/*
Matrix addition with to large matrices for the device memory, without utilizng streams.
*/

#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

// Grid size
#define B 100

// Block size
#define T 512

// Matrix dimension
#define C 51200L

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

int size = sizeof(int64_t);

// Host point64_ter
int64_t *a;
int64_t *b;
int64_t *c;

// Device point64_ter
int64_t *d_a;
int64_t *d_b;
int64_t *d_c;

// Host initialisation and matrix allocation
int init(){
    a = (int64_t *) malloc(C*C*size);
    b = (int64_t *) malloc(C*C*size);
    c = (int64_t *) malloc(C*C*size);
    for(int64_t i=0; i<C; i++){
        for(int64_t j=0; j<C; j++){
            b[j+i*C]=1;
            a[j+i*C]=1;
            c[j+i*C]=0;
        }
        
    }
        return 0;
}

// Kernel
__global__ void Mult(int64_t* d_a, int64_t* d_b, int64_t* d_c){
    int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    d_c[tid] = d_a[tid] + d_b[tid];
    
}

// Verifying results
int check(){
    bool test = false;
    for(int64_t i=0; i<C*C; i++){
        if(c[i]!=2){
            test = true;
        }
    }
    printf("Memory Access Issue visible: %s\n",test ? "true\n" : "false\n");
    return 0;
    
    
}

// Allocating device memory and copying matrices a and b from the host to d_a and d_b on the device
void initcuda(){
    
    cudaErrorCheck( hipMalloc(&d_a, C*C*size));
    cudaErrorCheck( hipMalloc(&d_b, C*C*size));
    cudaErrorCheck( hipMalloc(&d_c, C*C*size));
    cudaErrorCheck( hipMemcpy(d_a,a,C*C*size,hipMemcpyHostToDevice));
    cudaErrorCheck( hipMemcpy(d_b,b,C*C*size,hipMemcpyHostToDevice));
    
}

// Main program
int main(){
    // Initialisation
    init();
    initcuda();
    
    //Launch Kernel
    Mult<<<B,T>>>(d_a,d_b,d_c);
    
     // Check for errors in kernel launch (e.g. invalid execution configuration paramters)
    cudaErrorCheck( hipGetLastError());

    // Check for errors on the GPU after control is returned to CPU
    cudaErrorCheck( hipDeviceSynchronize());
    
    // Copying back the result d_c from the device to c on the host
    cudaErrorCheck( hipMemcpy(c,d_c,C*C*size,hipMemcpyHostToDevice));
    
    // Verifying results
    check();
    
    // Freeing device memory
    cudaErrorCheck( hipFree(d_a));
    cudaErrorCheck( hipFree(d_b));
    cudaErrorCheck( hipFree(d_c));
    
    // Freeing host memory
    free(a);
    free(b);
    free(c);
    return 0;
}